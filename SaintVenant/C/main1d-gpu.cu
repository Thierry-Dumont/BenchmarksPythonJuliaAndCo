#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <functional>
#include <chrono>
#include <fstream>

#include <Eigen/Dense>

#include "main1d-gpu-kernels.hpp"

struct Domain {
    constexpr Domain(double start, double end) : x_start(start), x_end(end) {}
    double x_start;
    double x_end;
};


template <typename Derived>
void init_solution(Eigen::ArrayBase<Derived>& V1, Eigen::ArrayBase<Derived>& V2, const Domain& domain) {
    const std::size_t nx = V1.size();
    const double hx = (domain.x_end - domain.x_start) / nx;
    const double xc = 0.5 * (domain.x_start + domain.x_end) + (nx / 8.0) * hx;

    for(std::size_t i=0; i < nx; ++i) {
        double x = domain.x_start + (i - 0.5) * hx;
        V1(i) = std::abs(x - xc) < 0.2 ? 1. : 0.;
        V2(i) = 0.;
    }           
}


template<std::size_t NUM_BLOCKS, std::size_t NUM_THREADS, std::size_t SMEM_SIZE, typename ArrayType>
double reduce(double* lambdas, double* lmb_max, std::size_t size, ArrayType& lmax) 
{
    // compute the maximum for each block
    reduce_max<<<NUM_BLOCKS, NUM_THREADS, SMEM_SIZE>>>(lambdas, lmb_max, size);

    // finishing on cpu. What if NUM_BLOCK >> NUM_THREADS ? Re-run a kernel ?
    hipMemcpy(lmax.data(), lmb_max, lmax.size() * sizeof(double), hipMemcpyDeviceToHost);
    return lmax.maxCoeff();
}



template<std::size_t NUM_BLOCKS, std::size_t NUM_THREADS, std::size_t NBLOCKS_MAX, std::size_t SMEM_MAX, typename ArrayType>
double update_to_time(double t, double tframe, double dx,
                      double* V1, double* V2,
                      double* Vold1, double* Vold2,
                      double* lambdas,
                      double* lmb_max, ArrayType& lmax,
                      double tol, std::size_t Nx)
{
    while(t < tframe) {
        update_eigenvalues<<<NUM_BLOCKS, NUM_THREADS>>>(lambdas, V1, V2, tol, Nx);
        double Cmax = reduce<NBLOCKS_MAX, NUM_THREADS, SMEM_MAX, ArrayType>(lambdas, lmb_max, Nx, lmax);
        double dt = std::min(0.5 * dx / Cmax, tframe - t);
        
        hipMemcpy(Vold1, V1, Nx * sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(Vold2, V2, Nx * sizeof(double), hipMemcpyDeviceToDevice);
        
        scheme_LaxFriedrich<<<NUM_BLOCKS, NUM_THREADS, 3*(NUM_THREADS + 2) * sizeof(double)>>>(V1, V2, Vold1, Vold2, lambdas, dt, dx, tol, Nx);
        
        t += dt;
    }

    return t;
}



int main() {
    constexpr Domain domain(0., 1.);
    constexpr std::size_t Nx = 32768;

    std::modulus<std::size_t> mod;
    
    constexpr std::size_t num_gpu_threads = 256;
    constexpr std::size_t num_gpu_blocks = mod(Nx, num_gpu_threads) == 0 ? Nx / num_gpu_threads : Nx / num_gpu_threads + 1;

    constexpr std::size_t nblocks_max = mod(num_gpu_blocks, 2) == 0 ? num_gpu_blocks / 2 : num_gpu_blocks / 2 + 1;
    constexpr std::size_t smem_max = num_gpu_threads <= 32 ? 64 * sizeof(double) : num_gpu_threads * sizeof(double);

    constexpr double T = 2.0;
    constexpr double dx = (domain.x_end - domain.x_start) / Nx;
    
    constexpr double tol = 1e-15;

    
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    
    
    Eigen::Array<double, 1, Eigen::Dynamic> lmax(1, nblocks_max);
    Eigen::Array<double, 1, Eigen::Dynamic> V1(1, Nx);
    Eigen::Array<double, 1, Eigen::Dynamic> V2(1, Nx);
    init_solution(V1, V2, domain);
    
    double *d_V1, *d_V2;
    hipMalloc((void**)&d_V1, Nx * sizeof(double));
    hipMalloc((void**)&d_V2, Nx * sizeof(double));
    
    hipMemcpy(d_V1, V1.data(), Nx * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V2, V2.data(), Nx * sizeof(double), hipMemcpyHostToDevice);
    
    double *d_Vold1, *d_Vold2;
    hipMalloc((void**)&d_Vold1, Nx * sizeof(double));
    hipMalloc((void**)&d_Vold2, Nx * sizeof(double));

    double* d_lambdas;
    hipMalloc((void**)&d_lambdas, Nx * sizeof(double));

    double* d_lmb_max;
    hipMalloc((void**)&d_lmb_max, nblocks_max * sizeof(double));


    
    double t = 0;
    std::cout << "Initial time t = " << t << std::endl;
    
    auto time_start = std::chrono::high_resolution_clock::now();
    t = update_to_time<num_gpu_blocks, num_gpu_threads, nblocks_max, smem_max>(t, T, dx,
                                                                               d_V1, d_V2,
                                                                               d_Vold1, d_Vold2,
                                                                               d_lambdas,
                                                                               d_lmb_max, lmax,
                                                                               tol, Nx);
    auto time_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = time_end - time_start;

    std::cout << "Elapsed time : " << elapsed_seconds.count() << "s" << std::endl;
    std::cout << "End of simulation, t = " << t << std::endl;


    // copy back the result and compute the mean value
    hipMemcpy(V1.data(), d_V1, Nx * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(V2.data(), d_V2, Nx * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << V1.mean() << std::endl;

    // write solution to file
    const Eigen::IOFormat to_file(Eigen::FullPrecision, Eigen::DontAlignCols, "\n", "\n");
    std::ofstream file("sol-gpu");
    file << V1.format(to_file);
    
    
    // freeing arrays on device
    hipFree(d_V1); hipFree(d_V2);
    hipFree(d_Vold1); hipFree(d_Vold2);
    hipFree(d_lambdas);
    hipFree(d_lmb_max);
    
    return 0;
}
