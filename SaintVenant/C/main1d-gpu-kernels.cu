#include "hip/hip_runtime.h"
#include "main1d-gpu-kernels.hpp"


__device__ inline double flux(const double V1, const double V2, const double tol) {
    if(fabs(V1) < tol)
        return 0.;
    else
        return V2 * V2 / V1 + 0.5 * 9.81 * V1 * V1;
}


__global__ void scheme_LaxFriedrich(double* V1, double* V2, double* Vold1, double* Vold2, double* lambdas, double dt, double dx, double tol, unsigned int nx) {
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double shared_data[];
    
    const unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int lid = threadIdx.x + 1;

    const double Cx = dt / dx;

    double* sVold1 = shared_data;
    double* sVold2 = &shared_data[blockDim.x + 2];
    double* sLmb = &shared_data[2 * (blockDim.x + 2)];

    if(gid < nx) {
        sVold1[lid] = Vold1[gid];
        sVold2[lid] = Vold2[gid];
        sLmb[lid] = lambdas[gid];

        if(lid == 1 && blockIdx.x > 0) {
            sVold1[0] = Vold1[gid - 1];
            sVold2[0] = Vold2[gid - 1];
            sLmb[0] = lambdas[gid - 1];
        }

        if(lid == blockDim.x && blockIdx.x < gridDim.x - 1) {
            sVold1[lid + 1] = Vold1[gid + 1];
            sVold2[lid + 1] = Vold2[gid + 1];
            sLmb[lid + 1] = lambdas[gid + 1];
        }
    }

    cg::sync(cta);


    if(gid > 0 && gid < nx-1) {
        double f_hu_m = flux(sVold1[lid-1], sVold2[lid-1], tol);
        double f_hu_p = flux(sVold1[lid+1], sVold2[lid+1], tol);
        double mlm = fmax(sLmb[lid-1], sLmb[lid]);
        double mlp = fmax(sLmb[lid], sLmb[lid+1]);

        V1[gid] += 0.5 * Cx * ((sVold2[lid-1] - sVold2[lid+1]) - mlm * (sVold1[lid] - sVold1[lid-1]) + mlp * (sVold1[lid+1] - sVold1[lid]));
        V2[gid] += 0.5 * Cx * ((f_hu_m - f_hu_p) - mlm * (sVold2[lid] - sVold2[lid-1]) + mlp * (sVold2[lid+1] - sVold2[lid]));
    }
    else if(gid == 0) {
        double f_hu_m = flux(sVold1[1], sVold2[1], tol);
        double f_hu_p = flux(sVold1[2], sVold2[2], tol);
        double ml = fmax(sLmb[1], sLmb[2]);

        V1[0] -= 0.5 * Cx * ((sVold2[2] + sVold2[1]) - ml * (sVold1[2] - sVold1[1]));
        V2[0] += Cx * (f_hu_m - sLmb[1] * sVold2[1]) - 0.5 * Cx * ((f_hu_p + f_hu_m) - ml * (sVold2[2] - sVold2[1]));
    }
    else if(gid == nx - 1) {
        double f_hu_m = flux(sVold1[lid-1], sVold2[lid-1], tol);
        double f_hu_p = flux(sVold1[lid], sVold2[lid], tol);
        double ml = fmax(sLmb[lid-1], sLmb[lid]);

        V1[nx-1] += 0.5 * Cx * ((sVold2[lid] + sVold2[lid-1]) - ml * (sVold1[lid] - sVold1[lid-1]));
        V2[nx-1] += 0.5 * Cx * ((f_hu_p + f_hu_m) - ml * (sVold2[lid] - sVold2[lid-1])) - Cx * (f_hu_p + sLmb[lid] * sVold2[lid]);
    }
}


__global__ void update_eigenvalues(double* lambdas, double* V1, double* V2, double tol, unsigned int nx) {
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < nx) {
        if(fabs(V1[index]) < tol)
            lambdas[index] = 0.;
        else
            lambdas[index] = fabs(V2[index] / V1[index]) + sqrt(9.81 * V1[index]);
    }
}




__global__ void reduce_max(double *in, double *out, std::size_t N) {
    cg::thread_block cta = cg::this_thread_block();
    extern __shared__ double shared_data[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    double mm = i < N ? in[i] : -1.0;     // because in this problem, the values of in are > 0
    if(i + blockDim.x < N)
        mm = fmax(mm, in[i + blockDim.x]);
    
    shared_data[tid] = mm;
    cg::sync(cta);

    // do reduction in shared mem
    if((blockDim.x >= 512) && (tid < 256)) {
        shared_data[tid] = mm = fmax(mm, shared_data[tid + 256]);
    }
    cg::sync(cta);

    if((blockDim.x >= 256) && (tid < 128)){
        shared_data[tid] = mm = fmax(mm, shared_data[tid + 128]);
    }
    cg::sync(cta);

    if((blockDim.x >= 128) && (tid <  64)){
       shared_data[tid] = mm = fmax(mm, shared_data[tid +  64]);
    }
    cg::sync(cta);

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);  // warp size
    if(cta.thread_rank() < 32) {
        // Fetch final intermediate max from 2nd warp
        if(blockDim.x >=  64) {
            mm = fmax(mm, shared_data[tid + 32]);
        }
        
        // Reduce final warp using shuffle
        for(int offset = tile32.size()/2; offset > 0; offset /= 2) {
            mm = fmax(mm, tile32.shfl_down(mm, offset));
        }
    }

    // write result for this block to global mem
    if(cta.thread_rank() == 0) {
        out[blockIdx.x] = mm;
    }
}